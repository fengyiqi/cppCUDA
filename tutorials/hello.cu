#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__global__ void helloFromGPU(void) {
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    // 目前我的nvcc 10.2还不支持核函数调用std函数
    printf("Hello from block %d and thread %d!\n", bid, tid);
}

class PRINT {
public:
    void print() {
        std::cout << "Hello from CPU" << std::endl;
    }
    void printGPU() {
        helloFromGPU<<<2, 4>>>();
    }
};

int main() {
    PRINT a;
    a.print();
    a.printGPU();
    hipDeviceSynchronize();
    // cudaDeviceReset();
    return 0;
}