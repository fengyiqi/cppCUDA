#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {
    const int grid_size = gridDim.x;
    const int block_size = blockDim.x;
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    printf("Grid size %d, block size %d. Hello from block %d and thread %d!\n", grid_size, block_size, bid, tid);
}

int main() {
    helloFromGPU<<<2, 4>>>();
    hipDeviceSynchronize();
    return 0;
}