#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "error.cuh"

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;

double __device__ add_element(const double &x, const double &y);
void __global__ add(const double *x, const double *y, double *z);
void check(const double *z, const int N);

int main() {
    const int N = 100000000;
    double *h_x = new double[N];
    double *h_y = new double[N];
    double *h_z = new double[N];

    for (int i = 0; i < N; i++) {
        h_x[i] = a;
        h_y[i] = b;
    }

    const int M = N * sizeof(double);
    double *d_x, *d_y, *d_z;
    CHECK(hipMalloc((void **)&d_x, M));
    CHECK(hipMalloc((void **)&d_y, M));
    CHECK(hipMalloc((void **)&d_z, M));
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice));

    const int block_size = 1280;
    const int grid_size = (N + block_size - 1) / block_size;
    add<<<grid_size, block_size>>>(d_x, d_y, d_z);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost));
    check(h_z, N);

    delete[] h_x, h_y, h_z;
    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipFree(d_z));

    return 0;
}

double __device__ add_element(const double &x, const double &y) {
    return x + y;
}

void __global__ add(const double *x, const double *y, double *z) {
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    // z[i] = add_element(x[i], y[i]);
    z[i] = x[i] + y[i];
}

void check(const double *z, const int N) { 
    for (int i = 0; i < N; i++) {
        if (fabs(z[i] - c) > EPSILON) {
            std::cout << "Has errors!\n";
            return;
        }
    }
    std::cout << "No errors!\n";
}