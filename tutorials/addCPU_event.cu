#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include "error.cuh"

#ifdef USE_DP
    typedef double real;
    const real EPSILON = 1.0e-15;
#else
    typedef float real;
    const real EPSILON = 1.0e-6f;
#endif

const int NUM_REPEATS = 11;
const real a = 1.23;
const real b = 2.34;
const real c = 3.57;

void add(const real *x, const real *y, real *z, const int N);
void check(const real *z, const int N);

int main() {
    const int N = 100000000;
    real *x = new real[N];
    real *y = new real[N];
    real *z = new real[N];

    for (int i = 0; i < N; i++) {
        x[i] = a;
        y[i] = b;
    }

    float t_sum = 0;
    float t2_sum = 0;
    int count = 0;
    for (int i = 0; i < NUM_REPEATS; i++) {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        add(x, y, z, N);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));

        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (i > 0) {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
            count += 1;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }
    
    const float t_ave = t_sum / count;
    const float t_err = sqrt(t2_sum / count - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);
    
    check(z, N);

    delete[] x, y, z;
    return 0;
}

void add(const real *x, const real *y, real *z, const int N) {
    for (int i = 0; i < N; i++)
        z[i] = x[i] + y[i];
}

void check(const real *z, const int N) { 
    for (int i = 0; i < N; i++) {
        if (fabs(z[i] - c) > EPSILON) {
            std::cout << "Has errors!\n";
            return;
        }
    }
    std::cout << "No errors!\n";
}