#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "error.cuh"

#ifdef USE_DP
    typedef double real;
    const real EPSILON = 1.0e-15;
#else
    typedef float real;
    const real EPSILON = 1.0e-6f;
#endif

const int NUM_REPEATS = 11;
const real a = 1.23;
const real b = 2.34;
const real c = 3.57;

void __global__ add(const real *x, const real *y, real *z);
void check(const real *z, const int N);

int main() {
    const int N = 100000000;
    real *h_x = new real[N];
    real *h_y = new real[N];
    real *h_z = new real[N];

    for (int i = 0; i < N; i++) {
        h_x[i] = a;
        h_y[i] = b;
    }

    const int M = N * sizeof(real);
    real *d_x, *d_y, *d_z;
    CHECK(hipMalloc((void **)&d_x, M));
    CHECK(hipMalloc((void **)&d_y, M));
    CHECK(hipMalloc((void **)&d_z, M));
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice));

    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size;

    float t_sum = 0;
    float t2_sum = 0;
    int count = 0;
    for (int i = 0; i < NUM_REPEATS; i++) {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        add<<<grid_size, block_size>>>(d_x, d_y, d_z);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));

        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Trial %d: time = %g ms.\n", count, elapsed_time);

        if (i > 0) {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
        count += 1;
    }
    const float t_ave = t_sum / (count - 1);
    const float t_err = sqrt(t2_sum / (count - 1) - t_ave * t_ave);
    printf("Average: time = %g +- %g ms.\n", t_ave, t_err);

    CHECK(hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost));
    check(h_z, N);

    delete[] h_x, h_y, h_z;
    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipFree(d_z));

    return 0;
}


void __global__ add(const real *x, const real *y, real *z) {
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    z[i] = x[i] + y[i];
}

void check(const real *z, const int N) { 
    for (int i = 0; i < N; i++) {
        if (fabs(z[i] - c) > EPSILON) {
            std::cout << "Has errors!\n";
            return;
        }
    }
    std::cout << "No errors!\n";
}