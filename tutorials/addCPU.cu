
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;

void add(const double *x, const double *y, double *z, const int N);
void check(const double *z, const int N);

int main() {
    const int N = 100000000;
    double *x = new double[N];
    double *y = new double[N];
    double *z = new double[N];

    for (int i = 0; i < N; i++) {
        x[i] = a;
        y[i] = b;
    }

    add(x, y, z, N);
    check(z, N);

    delete[] x, y, z;
    return 0;
}

void add(const double *x, const double *y, double *z, const int N) {
    for (int i = 0; i < N; i++)
        z[i] = x[i] + y[i];
}

void check(const double *z, const int N) { 
    for (int i = 0; i < N; i++) {
        if (fabs(z[i] - c) > EPSILON) {
            std::cout << "Has errors!\n";
            return;
        }
    }
    std::cout << "No errors!\n";
}